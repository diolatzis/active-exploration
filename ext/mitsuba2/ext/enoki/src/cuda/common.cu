#include "hip/hip_runtime.h"
/*
    src/cuda/common.cpp -- CUDA backend (wrapper routines)

    Enoki is a C++ template library that enables transparent vectorization
    of numerical kernels using SIMD instruction sets available on current
    processor architectures.

    Copyrighe (c) 2019 Wenzel Jakob <wenzel.jakob@epfl.ch>

    All rights reserved. Use of this source code is governed by a BSD-style
    license that can be found in the LICENSE file.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include "common.cuh"

NAMESPACE_BEGIN(enoki)

std::string mem_string(size_t size) {
    const char *orders[] = {
        "B", "KiB", "MiB", "GiB",
        "TiB", "PiB", "EiB"
    };
    float value = (float) size;

    int i = 0;
    for (i = 0; i < 6 && value >= 1024.f; ++i)
        value /= 1024.f;

    char buf[32];
    snprintf(buf, 32, "%.5g %s", value, orders[i]);

    return buf;
}

std::string time_string(size_t value_) {
    struct Order { float factor; const char* suffix; };
    const Order orders[] = { { 0, "us" },   { 1000, "ms" },
                             { 1000, "s" }, { 60, "m" },
                             { 60, "h" },   { 24, "d" },
                             { 7, "w" },    { (float) 52.1429, "y" } };

    int i = 0;
    float value = (float) value_;
    for (i = 0; i < 7 && value > orders[i+1].factor; ++i)
        value /= orders[i+1].factor;

    char buf[32];
    snprintf(buf, 32, "%.5g %s", value, orders[i].suffix);

    return buf;
}

template <typename T> __global__ void fill(T *out, T value, size_t n) {
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n;
         i += blockDim.x * gridDim.x)
        out[i] = value;
}

template <typename T> __global__ void set_value(T *ptr, size_t idx, T value) {
    ptr[idx] = value;
}

ENOKI_EXPORT void cuda_fill(uint8_t *ptr, uint8_t value, size_t size) {
    cuda_check(hipMemsetAsync(ptr, value, size));
}

ENOKI_EXPORT void cuda_fill(uint16_t *ptr, uint16_t value, size_t size) {
    fill<<<256, 256>>>(ptr, value, size);
}

ENOKI_EXPORT void cuda_fill(uint32_t *ptr, uint32_t value, size_t size) {
    fill<<<256, 256>>>(ptr, value, size);
}

ENOKI_EXPORT void cuda_fill(uint64_t *ptr, uint64_t value, size_t size) {
    fill<<<256, 256>>>(ptr, value, size);
}

template <typename T> __global__ void reverse(T *out, const T *in, size_t n) {
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n;
         i += blockDim.x * gridDim.x)
        out[i] = in[n - 1 - i];
}

ENOKI_EXPORT void cuda_reverse(uint8_t *out, const uint8_t *in, size_t size) {
    reverse<<<256, 256>>>(out, in, size);
}

ENOKI_EXPORT void cuda_reverse(uint16_t *out, const uint16_t *in, size_t size) {
    reverse<<<256, 256>>>(out, in, size);
}

ENOKI_EXPORT void cuda_reverse(uint32_t *out, const uint32_t *in, size_t size) {
    reverse<<<256, 256>>>(out, in, size);
}

ENOKI_EXPORT void cuda_reverse(uint64_t *out, const uint64_t *in, size_t size) {
    reverse<<<256, 256>>>(out, in, size);
}

ENOKI_EXPORT void cuda_memcpy_to_device(void *dst, const void *src, size_t size) {
    cuda_check(hipMemcpy(dst, src, size, hipMemcpyHostToDevice));
}

ENOKI_EXPORT void cuda_memcpy_from_device(void *dst, const void *src, size_t size) {
    cuda_check(hipMemcpy(dst, src, size, hipMemcpyDeviceToHost));
}

ENOKI_EXPORT void cuda_memcpy_to_device_async(void *dst, const void *src, size_t size) {
    cuda_check(hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice));
}

ENOKI_EXPORT void cuda_memcpy_from_device_async(void *dst, const void *src, size_t size) {
    cuda_check(hipMemcpyAsync(dst, src, size, hipMemcpyDeviceToHost));
}

ENOKI_EXPORT void cuda_mem_get_info(size_t *free, size_t *total) {
    cuda_check(hipMemGetInfo(free, total));
}

struct CUDAErrorList {
    hipError_t id;
    const char *value;
};

static CUDAErrorList __cuda_error_list[] = {
    { hipSuccess,
     "hipSuccess"},
    { hipErrorInvalidValue,
     "hipErrorInvalidValue"},
    { hipErrorOutOfMemory,
     "hipErrorOutOfMemory"},
    { hipErrorNotInitialized,
     "hipErrorNotInitialized"},
    { hipErrorDeinitialized,
     "hipErrorDeinitialized"},
    { hipErrorProfilerDisabled,
     "hipErrorProfilerDisabled"},
    { hipErrorProfilerNotInitialized,
     "hipErrorProfilerNotInitialized"},
    { hipErrorProfilerAlreadyStarted,
     "hipErrorProfilerAlreadyStarted"},
    { hipErrorProfilerAlreadyStopped,
     "hipErrorProfilerAlreadyStopped"},
    { hipErrorNoDevice,
     "hipErrorNoDevice"},
    { hipErrorInvalidDevice,
     "hipErrorInvalidDevice"},
    { hipErrorInvalidImage,
     "hipErrorInvalidImage"},
    { hipErrorInvalidContext,
     "hipErrorInvalidContext"},
    { hipErrorContextAlreadyCurrent,
     "hipErrorContextAlreadyCurrent"},
    { hipErrorMapFailed,
     "hipErrorMapFailed"},
    { hipErrorUnmapFailed,
     "hipErrorUnmapFailed"},
    { hipErrorArrayIsMapped,
     "hipErrorArrayIsMapped"},
    { hipErrorAlreadyMapped,
     "hipErrorAlreadyMapped"},
    { hipErrorNoBinaryForGpu,
     "hipErrorNoBinaryForGpu"},
    { hipErrorAlreadyAcquired,
     "hipErrorAlreadyAcquired"},
    { hipErrorNotMapped,
     "hipErrorNotMapped"},
    { hipErrorNotMappedAsArray,
     "hipErrorNotMappedAsArray"},
    { hipErrorNotMappedAsPointer,
     "hipErrorNotMappedAsPointer"},
    { hipErrorECCNotCorrectable,
     "hipErrorECCNotCorrectable"},
    { hipErrorUnsupportedLimit,
     "hipErrorUnsupportedLimit"},
    { hipErrorContextAlreadyInUse,
     "hipErrorContextAlreadyInUse"},
    { hipErrorPeerAccessUnsupported,
     "hipErrorPeerAccessUnsupported"},
    { hipErrorInvalidKernelFile,
     "hipErrorInvalidKernelFile"},
    { hipErrorInvalidGraphicsContext,
     "hipErrorInvalidGraphicsContext"},
    { CUDA_ERROR_NVLINK_UNCORRECTABLE,
     "CUDA_ERROR_NVLINK_UNCORRECTABLE"},
    { CUDA_ERROR_JIT_COMPILER_NOT_FOUND,
     "CUDA_ERROR_JIT_COMPILER_NOT_FOUND"},
    { hipErrorInvalidSource,
     "hipErrorInvalidSource"},
    { hipErrorFileNotFound,
     "hipErrorFileNotFound"},
    { hipErrorSharedObjectSymbolNotFound,
     "hipErrorSharedObjectSymbolNotFound"},
    { hipErrorSharedObjectInitFailed,
     "hipErrorSharedObjectInitFailed"},
    { hipErrorOperatingSystem,
     "hipErrorOperatingSystem"},
    { hipErrorInvalidHandle,
     "hipErrorInvalidHandle"},
    { hipErrorNotFound,
     "hipErrorNotFound"},
    { hipErrorNotReady,
     "hipErrorNotReady"},
    { hipErrorIllegalAddress,
     "hipErrorIllegalAddress"},
    { hipErrorLaunchOutOfResources,
     "hipErrorLaunchOutOfResources"},
    { hipErrorLaunchTimeOut,
     "hipErrorLaunchTimeOut"},
    { CUDA_ERROR_LAUNCH_INCOMPATIBLE_TEXTURING,
     "CUDA_ERROR_LAUNCH_INCOMPATIBLE_TEXTURING"},
    { hipErrorPeerAccessAlreadyEnabled,
     "hipErrorPeerAccessAlreadyEnabled"},
    { hipErrorPeerAccessNotEnabled,
     "hipErrorPeerAccessNotEnabled"},
    { hipErrorSetOnActiveProcess,
     "hipErrorSetOnActiveProcess"},
    { hipErrorContextIsDestroyed,
     "hipErrorContextIsDestroyed"},
    { hipErrorAssert,
     "hipErrorAssert"},
    { CUDA_ERROR_TOO_MANY_PEERS,
     "CUDA_ERROR_TOO_MANY_PEERS"},
    { hipErrorHostMemoryAlreadyRegistered,
     "hipErrorHostMemoryAlreadyRegistered"},
    { hipErrorHostMemoryNotRegistered,
     "hipErrorHostMemoryNotRegistered"},
    { CUDA_ERROR_HARDWARE_STACK_ERROR,
     "CUDA_ERROR_HARDWARE_STACK_ERROR"},
    { CUDA_ERROR_ILLEGAL_INSTRUCTION,
     "CUDA_ERROR_ILLEGAL_INSTRUCTION"},
    { CUDA_ERROR_MISALIGNED_ADDRESS,
     "CUDA_ERROR_MISALIGNED_ADDRESS"},
    { CUDA_ERROR_INVALID_ADDRESS_SPACE,
     "CUDA_ERROR_INVALID_ADDRESS_SPACE"},
    { CUDA_ERROR_INVALID_PC,
     "CUDA_ERROR_INVALID_PC"},
    { hipErrorLaunchFailure,
     "hipErrorLaunchFailure"},
    { hipErrorCooperativeLaunchTooLarge,
     "hipErrorCooperativeLaunchTooLarge"},
    { CUDA_ERROR_NOT_PERMITTED,
     "CUDA_ERROR_NOT_PERMITTED"},
    { hipErrorNotSupported,
     "hipErrorNotSupported"},
    { hipErrorUnknown,
     "hipErrorUnknown"},
    { (hipError_t) -1, nullptr }
};

ENOKI_EXPORT const char *cuda_error_string(hipError_t id) {
    int index = 0;

    while (__cuda_error_list[index].id != id &&
           __cuda_error_list[index].id != (hipError_t) -1)
        index++;

    if (__cuda_error_list[index].id == id)
        return __cuda_error_list[index].value;
    else
        return "Invalid CUDA error status!";
}

ENOKI_EXPORT void cuda_check_impl(hipError_t errval, const char *file, const int line) {
    if (errval != hipSuccess && errval != hipErrorDeinitialized) {
        const char *err_msg = cuda_error_string(errval);
        fprintf(stderr,
                "cuda_check(): driver API error = %04d \"%s\" in "
                "%s:%i.\n", (int) errval, err_msg, file, line);
        exit(EXIT_FAILURE);
    }
}

ENOKI_EXPORT void cuda_check_impl(hipError_t errval, const char *file, const int line) {
    if (errval != hipSuccess && errval != hipErrorDeinitialized) {
        const char *err_msg = hipGetErrorName(errval);
        fprintf(stderr,
                "cuda_check(): runtime API error = %04d \"%s\" in "
                "%s:%i.\n", (int) errval, err_msg, file, line);
        exit(EXIT_FAILURE);
    }
}

NAMESPACE_END(enoki)
